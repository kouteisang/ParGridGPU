#include "Graph/MultilayerGraph.h"
#include "utils.h"
#include "Algorithm/ParPeel.cuh"
#include "Algorithm/ParPeel_klist.cuh"




enum Algorithm{
    llist = 1,
    klist = 2,
};

int main(int argc, char* argv[]){

    string dataset = "example";
    int order = 0;
    int alg = 1;

    for(int i = 1; i < argc; i ++){
        string arg = argv[i];
        if (arg == "-d" && i + 1 < argc) {
            dataset = argv[++i];
        }else if(arg == "-a" && i+1 < argc){
            alg = std::stoi(argv[++i]);
        }
    }


    // load the dataset
    MultilayerGraph mg;
    // relative path is relative to the execute file
    mg.LoadFromFile("./dataset/"+dataset+"/");
    mg.SetGraphOrder(0);
    uint *order_list = mg.GetOrder();
    for(int i = 0; i < mg.getLayerNumber(); i ++){
        cout << order_list[i] << " ";;
    }
    cout << endl;
    mg.PrintStatistics();

    int n_vertex = mg.GetN();
    int n_layer = mg.getLayerNumber();

    int *degs;
    degs = new int[n_vertex * n_layer];
    for(int v = 0; v < n_vertex; v ++){
        for(int l = 0; l < n_layer; l ++){
            degs[v * n_layer + l] = mg.GetGraph(l).GetAdjLst()[v][0];
        }
    }
   

    G_pointers data_pointers;
    data_pointers.n_vtx = n_vertex;
    data_pointers.n_layer = n_layer;

    // // memory alloc adj list for each layer
    std::vector<int*> d_adj_list_ptrs(n_layer);

    int* h_offset;
    h_offset = new int[n_layer * (n_vertex+1)];
    int cnt = 0;

    int *num_edge;
    num_edge = new int[n_layer];

    for(int l = 0; l < n_layer; l ++){

        std::vector<int> h_adj_list;
        int totalsum = 0;
        h_offset[cnt ++] = totalsum;

        uint** adj_lst = mg.GetGraph(l).GetAdjLst();
        for(int v = 0; v < n_vertex; v ++){
            int offset = mg.GetGraph(l).GetAdjLst()[v][0];
            totalsum += offset;
            h_offset[cnt ++] = totalsum;
            for(int nb = 1; nb <= offset; nb ++){
                h_adj_list.push_back(adj_lst[v][nb]);
            }
        }
        
        // for(int uu = 0; uu < h_adj_list.size(); uu ++){
        //     cout << h_adj_list[uu] << " ";
        // }

        int* d_list;
        int len = h_adj_list.size();
        num_edge[l] = len;
        chkerr(hipMalloc(&d_list, len * sizeof(int)));
        hipMemcpy(d_list, h_adj_list.data(), sizeof(int) * len, hipMemcpyHostToDevice);
        d_adj_list_ptrs[l] = d_list;
    }



    malloc_graph_gpu_memory(data_pointers, degs, h_offset, d_adj_list_ptrs);

    hipEvent_t start, stop; // Calculate time
    hipEventCreate(&start); // Calculate time
    hipEventCreate(&stop);  // Calculate time
    
    hipEventRecord(start, 0);

    switch (alg)
    {
        case Algorithm::llist:
            gpu_baseline_de(data_pointers, degs);
            break;
        case Algorithm::klist:
            gpu_baseline_de_klist(data_pointers, degs);
            break;
        default:
            break;
    }

    



    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, stop);
    std::cout << "GPU time = " << gpu_time << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);


    // ll_uint *id2vtx = new ll_uint[mg.GetN()];
    // mg.LoadId2VtxMap(id2vtx);



}