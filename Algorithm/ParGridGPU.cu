#include "hip/hip_runtime.h"
#include "ParGridGPU.cuh"

__global__ void trykernel(int** adj ,int* deg, int* offset, int n_layer, int n_vtx){

    // printf("Hello world\n");
    // for(int v = 0; v < n_vtx; v ++){
    //     for(int l = 0; l < n_layer; l ++){
    //         printf("%d ,", deg[v*n_layer+l]);
    //     }
    //     printf("\n\n");
    // }
    // for(int l = 0; l < n_layer; l ++){
    //     for(int v = 0; v < n_vtx; v ++){
    //         int begin = l * (n_vtx+1) + v;
    //         int end =  l * (n_vtx+1) + v + 1;
    //         printf("%d, %d\n", offset[begin], offset[end]);
    //     }
    // }

    // printf("\n");
    // printf("\n");
    // for(int l = 0; l  < n_layer; l ++){
    //     int len = offset[(n_vtx+1)*(l+1) - 1];
    //     // printf("len = %d", len);
    //     int* adj_l = adj[l];
    //     for(int v = 0; v < len; v ++){
    //         printf("%d,", adj_l[v]);
    //     }
    //     printf("\n");
    // }

    // // for(int l = 0; l  < n_layer; l ++){
    //     int len = 18;
    //     int* adj_l = adj[0];
    //     for(int v = 0; v < 18; v ++){
    //         printf("%d,", adj_l[v]);
    //     }
    //     printf("\n\n\n");
    // // }

}


__global__ void scan(int* global_buffer, int* buf_count, int* deg, int* precount, int* visit, int n_vtx, int n_layer, int k, int l){
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int sh_buf_count;
    __shared__ int* t_global_buffer;

    if(threadIdx.x == 0){
        sh_buf_count = 0;
        t_global_buffer = global_buffer + blockIdx.x * BUFFER_SIZE;
    }
    __syncthreads();

    for(int v = tid; v < n_vtx; v += BLK_DIM * BLK_NUMS){
        if(visit[v] == 1) continue;
        int count = 0;
        int start = v*n_layer;
        int end = (v+1)*n_layer;
        for(int d = start; d < end; d ++){
            count += (deg[d] >= k);
        }
        if(count < l){
            visit[v] = 1; // visit 设置为false
            count = 0;  // count 设置为 0
            int pos = atomicAdd(&sh_buf_count, 1);
            t_global_buffer[pos] = v;
            // printf("v = %d\n", v);
        }
        precount[v] = count;
    }
    __syncthreads();

    if(threadIdx.x == 0){
        buf_count[blockIdx.x] = sh_buf_count;
    
    }
}

__global__ void update(int* global_buffer, int* buf_count, int *deg, int **adj, int* offset, int* precount, int* visit, int n_vtx, int n_layer, int k, int lambda, int* global_count){
    
    __shared__ int start, end;
    __shared__ int* t_global_buffer;

    int warp_per_block = blockDim.x / WARP_SIZE;
    int warp_id = threadIdx.x / WARP_SIZE;
    int lane_id = threadIdx.x % WARP_SIZE;
    int start_prime, end_prime;
    if(threadIdx.x == 0){
        t_global_buffer = global_buffer + blockIdx.x * BUFFER_SIZE;
        start = 0;
        end = buf_count[blockIdx.x]; // The end position of the buffer
        // printf("id = %d, end = %d\n", blockIdx.x, end);
    } 

    __syncthreads();

    while(true){
        __syncthreads();
        // printf("end = %d\n", end);
        if(start >= end) break; // All the thread break the iteration
        start_prime = start + warp_id; // Get the vertex id position
        end_prime = end; // Get the last position of the vertex id
        __syncthreads();
        if(start_prime >= end_prime) continue; // The vertex position is larger than the number of valid vertices in the buffer
        if(threadIdx.x == 0){
            start = min(start + warp_per_block, end); // update the start position
        }
        int v = t_global_buffer[start_prime]; // Get the vertex id

        for(int l = 0; l < n_layer; l ++){
            __syncwarp();
            int offset_start = offset[l * (n_vtx+1) + v]; // offset of v 
            int offset_end = offset[l * (n_vtx+1) + v + 1]; // offset of v
            int* adj_l = adj[l];
            while (true){
                __syncwarp();
                if(offset_start >= offset_end) break;
                int uid = offset_start + lane_id;
                offset_start = offset_start + WARP_SIZE; // update the offset position, each thread maintain its own offset_start
                if(uid >= offset_end) continue; // This vertex does not has so many neighbouthood
                int u = adj_l[uid]; // v's out-neighbouthood u
                if(visit[u] == 1) continue;
                // printf("l = %d, v = %d, u = %d\n", l, v, u);
                int originDeg = atomicSub(&deg[u*n_layer + l], 1);
                if (originDeg == k){
                    // printf("u = %d, originDeg = %d, precount = %d\n", u, originDeg, precount[u]);
                    int originCnt = atomicSub(&precount[u], 1);
                    // printf("originCnt = %d\n", originCnt); 
                    if(originCnt == lambda && visit[u] == 0){
                        // printf("u = %d\n", u);
                        visit[u] = 1;
                        precount[u] = 0;
                        int end_pos = atomicAdd(&end, 1);
                        t_global_buffer[end_pos] = u;
                    }
                }
            }

        }
    }

    if(threadIdx.x == 0 && end > 0){
        atomicAdd(global_count, end);
    }


}

void gpu_baseline_de(G_pointers &p, int* dges){
    // printf("Here?\n");
    // trykernel<<<1, 1>>>(p.adj, p.deg, p.offset, p.n_layer, p.n_vtx);
    // hipDeviceSynchronize(); 

    int* global_count = 0;
    chkerr(hipMalloc(&global_count, sizeof(int)));

    int* buf_count;
    chkerr(hipMalloc(&buf_count, sizeof(int) * BLK_NUMS));
    hipMemset(buf_count, 0, sizeof(int) * BLK_NUMS);

    int* global_buffer;
    chkerr(hipMalloc(&global_buffer, sizeof(int) * BLK_NUMS * BUFFER_SIZE));
    

    int n_layer = p.n_layer;
    int n_vtx = p.n_vtx;

    int k = 0;
    int count = 0;

    for(int l = 1; l <= n_layer; l ++ ){
        k = 1;
        hipMemset(global_count, 0, sizeof(int));
        hipMemset(p.precount, 0, sizeof(int)*n_vtx);
        hipMemset(p.visit, 0, p.n_vtx * sizeof(int)); // flag = false means has not visited
        chkerr(hipMemcpy(p.t_deg, p.deg, p.n_vtx * p.n_layer * sizeof(int), hipMemcpyDeviceToDevice));
        count = 0;
        while(count < n_vtx){ 
            hipMemset(buf_count, 0, sizeof(int) * BLK_NUMS);
            scan<<<BLK_NUMS, BLK_DIM>>>(global_buffer, buf_count, p.t_deg, p.precount, p.visit,n_vtx, n_layer, k, l);
            update<<<BLK_NUMS, BLK_DIM>>>(global_buffer, buf_count, p.t_deg, p.adj, p.offset, p.precount, p.visit, n_vtx, n_layer, k, l, global_count);
            chkerr(hipMemcpy(&count, global_count, sizeof(int), hipMemcpyDeviceToHost));
            // printf("count = %d\n", count);
            // printf("l = %d, k = %d, valid = %d\n", l, k, n_vtx - count);
            if(count < n_vtx){
                k ++;
            }else if(count >= n_vtx){
               break;
            }
        }
        if(k == 1 && count == n_vtx) break;
    }
}

// 0 1 2
// 0 3 5